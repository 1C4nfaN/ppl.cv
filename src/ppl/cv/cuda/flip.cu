#include "hip/hip_runtime.h"
/**
 * @file   flip.cu
 * @brief  The kernel and invocation definitions of image flipping operation.
 * @author Liheng Jian(jianliheng@sensetime.com)
 *
 * @copyright Copyright (c) 2014-2021 SenseTime Group Limited.
 */

#include "flip.h"

#include <cassert>

#include "utility.hpp"

using namespace ppl::common;

namespace ppl {
namespace cv {
namespace cuda {

template <typename T0, typename T1>
__global__
void flipKernel(const T1* src, int rows, int cols, int src_stride, T1* dst,
                int dst_stride, int flip_code) {
  int element_x = (blockIdx.x << kBlockShiftX1) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  T0* output = (T0*)((uchar*)dst + element_y * dst_stride);

  int x, y;
  if (flip_code == 0) {
    x = element_x;
    y = rows - element_y - 1;
  }
  else if (flip_code > 0) {
    x = cols - element_x - 1;
    y = element_y;
  }
  else {
    x = cols - element_x - 1;
    y = rows - element_y - 1;
  }
  T0* input = (T0*)((uchar*)src + y * src_stride);

  if (element_x < cols && element_y < rows) {
    T0 result = input[x];
    output[element_x] = result;
  }
}

RetCode flip(const uchar* src, int rows, int cols, int channels, int src_stride,
             uchar* dst, int dst_stride, int flip_code, hipStream_t stream) {
  if (src == nullptr || dst == nullptr || rows < 1 || cols < 1 ||
      (channels != 1 && channels != 3 && channels != 4) ||
      src_stride < cols * channels * sizeof(uchar) ||
      dst_stride < cols * channels * sizeof(uchar)) {
    return RC_INVALID_VALUE;
  }

  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  if (channels == 1) {
    flipKernel<uchar, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 3) {
    flipKernel<uchar3, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 4) {
    flipKernel<uchar4, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else {
  }

  return RC_SUCCESS;
}

RetCode flip(const float* src, int rows, int cols, int channels, int src_stride,
             float* dst, int dst_stride, int flip_code, hipStream_t stream) {
  if (src == nullptr || dst == nullptr || rows < 1 || cols < 1 ||
      (channels != 1 && channels != 3 && channels != 4) ||
      src_stride < cols * channels * sizeof(float) ||
      dst_stride < cols * channels * sizeof(float)) {
    return RC_INVALID_VALUE;
  }

  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  if (channels == 1) {
    flipKernel<float, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 3) {
    flipKernel<float3, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 4) {
    flipKernel<float4, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else {
  }

  return RC_SUCCESS;
}

template <>
RetCode Flip<uchar, 1>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const uchar* inData,
                       int outWidthStride,
                       uchar* outData,
                       int flipCode) {
  RetCode code = flip(inData, height, width, 1, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<uchar, 3>(hipStream_t stream,
                   int height,
                   int width,
                   int inWidthStride,
                   const uchar* inData,
                   int outWidthStride,
                   uchar* outData,
                   int flipCode) {
  RetCode code = flip(inData, height, width, 3, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<uchar, 4>(hipStream_t stream,
                   int height,
                   int width,
                   int inWidthStride,
                   const uchar* inData,
                   int outWidthStride,
                   uchar* outData,
                   int flipCode) {
  RetCode code = flip(inData, height, width, 4, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 1>(hipStream_t stream,
                    int height,
                    int width,
                    int inWidthStride,
                    const float* inData,
                    int outWidthStride,
                    float* outData,
                    int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 1, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 3>(hipStream_t stream,
                    int height,
                    int width,
                    int inWidthStride,
                    const float* inData,
                    int outWidthStride,
                    float* outData,
                    int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 3, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 4>(hipStream_t stream,
                    int height,
                    int width,
                    int inWidthStride,
                    const float* inData,
                    int outWidthStride,
                    float* outData,
                    int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 4, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

}  // namespace cuda
}  // namespace cv
}  // namespace ppl
